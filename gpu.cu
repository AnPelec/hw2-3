#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/scan.h>
#include <thrust/copy.h>
#include <iostream>

#include <algorithm>

#define NUM_THREADS 256

// Put any static global variables here that you will use throughout the simulation.
int blks;

int grid_side_length;       // number of buckets on each side
int num_buckets;            // total number of buckets

int* bucket_sizes;          // show where each bucket starts and ends
int* bucket_index;          // used to place each particle at a unique index in each bucket

int* particles_in_buckets;  // stores the particle indices according to their buckets
                            // has size 2 * num_parts since we need extra space for the rebucketing
int cnt;                    // whether we are using the first or the second half of particles_in_buckets

// This function computes the bucket of a particle
__device__ void particle_to_bucket(particle_t particle, double size, int grid_side_length, int &bx, int &by) {
    bx = (particle.x * grid_side_length)/size;
    by = (particle.y * grid_side_length)/size;

    // Sanity checks
    assert(0 <= bx && bx < grid_side_length);
    assert(0 <= by && by < grid_side_length);
}

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

__global__ void compute_forces_gpu(particle_t* parts, int num_parts, int* particles_in_buckets, double size, int grid_side_length, int cnt, int* bucket_sizes) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < num_parts; i += stride) {
        int particle_index = particles_in_buckets[cnt * num_parts + i];

        parts[particle_index].ax = parts[particle_index].ay = 0;

        // find nearby buckets
        int bucket_row, bucket_col;
        particle_to_bucket(parts[particle_index], size, grid_side_length, bucket_row, bucket_col);
        
        for (int bx = max(bucket_row-1, 0); bx <= min(bucket_row+1, grid_side_length-1); bx ++) {
            for (int by = max(bucket_col-1, 0); by <= min(bucket_col+1, grid_side_length-1); by ++) {
                
                int neighbor_bucket = bx * grid_side_length + by;
                
                int start_index, end_index;
                if (neighbor_bucket == 0) {
                    start_index = 0;
                } else {
                    start_index = bucket_sizes[neighbor_bucket-1];
                }
                end_index = bucket_sizes[neighbor_bucket];

                for (int j = start_index; j < end_index; ++ j) {
                    int neighbor_index = particles_in_buckets[cnt*num_parts + j];

                    apply_force_gpu(parts[particle_index], parts[neighbor_index]);
                }
            }
        }
    }
}

__global__ void move_gpu(particle_t* parts, int num_parts, int* particles_in_buckets, double size, int cnt) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < num_parts; i += stride) {
        int particle_index = particles_in_buckets[cnt*num_parts + i];

        particle_t* p = &parts[particle_index];
        //
        //  slightly simplified Velocity Verlet integration
        //  conserves energy better than explicit Euler method
        //
        p->vx += p->ax * dt;
        p->vy += p->ay * dt;
        p->x += p->vx * dt;
        p->y += p->vy * dt;

        //
        //  bounce from walls
        //
        while (p->x < 0 || p->x > size) {
            p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
            p->vx = -(p->vx);
        }
        while (p->y < 0 || p->y > size) {
            p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
            p->vy = -(p->vy);
        }
    }
}

/*
    HELPER FUNCTIONS
*/

__global__ void compute_bucket_sizes(particle_t* parts, int num_parts, int* particles_in_buckets, int cnt, int* bucket_sizes, double size, int grid_side_length) { 
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < num_parts; i += stride) {
        int particle_index = particles_in_buckets[cnt*num_parts + i];
        if (particle_index == -1) { // for the first
            particle_index = i;
        }

        // compute which bucket you are in
        int bucket_row, bucket_col;
        particle_to_bucket(parts[particle_index], size, grid_side_length, bucket_row, bucket_col);
        int current_bucket = bucket_row * grid_side_length + bucket_col;

        // increase the size of this bucket (atomically)
        // the second parameter is used if we want to take modulo something
        auto old_value = atomicAdd(bucket_sizes + current_bucket, 1);
    }
}

__global__ void rebucket_particles(particle_t* parts, int num_parts, int* particles_in_buckets, int cnt, int* bucket_sizes, double size, int grid_side_length, int* bucket_index) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < num_parts; i += stride) {
        int particle_index = particles_in_buckets[cnt*num_parts + i];
        if (particle_index == -1) { // for the first time
            particle_index = i;
        }

        // compute which bucket you are in
        int bucket_row, bucket_col;
        particle_to_bucket(parts[particle_index], size, grid_side_length, bucket_row, bucket_col);
        int current_bucket = bucket_row * grid_side_length + bucket_col;

        // obtain your index in the bucket
        auto particle_index_in_bucket = atomicAdd(bucket_index + current_bucket, 1);

        int bucket_offset = 0;
        if (current_bucket > 0) {
            bucket_offset = bucket_sizes[current_bucket-1];
        }

        // move to the new bucket
        particles_in_buckets[(1-cnt)*num_parts + bucket_offset + particle_index_in_bucket] = particle_index;
    }
} 


/*
    MAIN SIMULATION
*/

void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here

    // I think we can adjust this parameter
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;

    grid_side_length = std::min(int(size/(2*cutoff)), int(sqrt(num_parts))); // Number of rows/columns in our grid of buckets
    num_buckets = grid_side_length*grid_side_length;

    // 1. Allocate space on the GPU
    cnt = 0;
    hipMalloc((void **)&particles_in_buckets, 2 * num_parts * sizeof(int));
    hipMalloc((void **)&bucket_sizes, num_buckets * sizeof(int));
    hipMalloc((void **)&bucket_index, num_buckets * sizeof(int));

    // 2. Move particles to the GPU (the first time they point to something empty)
    hipMemset(particles_in_buckets, -1, num_parts * sizeof(int));

    // 3. Initialize the buckets
    hipMemset(bucket_sizes, 0, num_buckets * sizeof(int)); // zero out bucket sizes
    // 3a. Compute bucket sizes
    compute_bucket_sizes<<<blks, NUM_THREADS>>>(parts, num_parts, particles_in_buckets, cnt, bucket_sizes, size, grid_side_length);
    // 3b. Inclusive scan for indices
    thrust::device_ptr<int> bucket_sizes_ptr = thrust::device_pointer_cast(bucket_sizes);
    thrust::inclusive_scan(bucket_sizes_ptr,
                           bucket_sizes_ptr + num_buckets, 
                           bucket_sizes_ptr);
    // 3c. Zero out bucket index
    hipMemset(bucket_index, 0, num_buckets * sizeof(int));
    // 3d. Move particles to the correct bucket
    rebucket_particles<<<blks, NUM_THREADS>>>(parts, num_parts, particles_in_buckets, cnt, bucket_sizes, size, grid_side_length, bucket_index);
    // set cnt to 1-cnt
    cnt = 1 - cnt;
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    // parts live in GPU memory
    // Rewrite this function

    // Step 1. Compute forces
    compute_forces_gpu<<<blks, NUM_THREADS>>>(
        parts, num_parts, particles_in_buckets, size, grid_side_length, cnt, bucket_sizes
    );

    // Step 2. Move particles
    move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, particles_in_buckets, size, cnt);

    // Step 3. Compute new bucket sizes
    hipMemset(bucket_sizes, 0, num_buckets * sizeof(int)); // zero out current sizes
    compute_bucket_sizes<<<blks, NUM_THREADS>>>(parts, num_parts, particles_in_buckets, cnt, bucket_sizes, size, grid_side_length); // actual bucket size computation
    // inclusive scan
    thrust::device_ptr<int> bucket_sizes_ptr = thrust::device_pointer_cast(bucket_sizes);
    thrust::inclusive_scan(bucket_sizes_ptr, 
                           bucket_sizes_ptr + num_buckets, 
                           bucket_sizes_ptr);
    // 3c. Zero out bucket index
    hipMemset(bucket_index, 0, num_buckets * sizeof(int));

    // Step 4. Rebucket particles
    rebucket_particles<<<blks, NUM_THREADS>>>(parts, num_parts, particles_in_buckets, cnt, bucket_sizes, size, grid_side_length, bucket_index);

    // set cnt to 1-cnt
    cnt = 1 - cnt;
}
